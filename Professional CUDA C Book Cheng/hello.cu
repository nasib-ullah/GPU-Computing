
#include <hip/hip_runtime.h>
#include<stdio.h>


__global__ void hellofromGPU(void)
{
   int tid = threadIdx.x;
   printf("%d",tid);
   printf("Hello world from GPU\n");
}

int main(void)
{
  printf("Hello world from CPU \n ");
  hellofromGPU<<<1,10>>>();
  //cudaDeviceReset();
  return 0;
}
